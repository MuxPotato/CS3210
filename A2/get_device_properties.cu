#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    // Get the number of CUDA devices
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found!" << std::endl;
        return 1;
    }

    // Loop through all devices
    for (int device = 0; device < deviceCount; device++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x " 
                  << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << std::endl;
        std::cout << "  Max grid dimensions: " << deviceProp.maxGridSize[0] << " x " 
                  << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << std::endl;
        std::cout << "  Shared memory per block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Number of SMs: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Warp size: " << deviceProp.warpSize << std::endl;
        std::cout << "  Memory clock rate: " << deviceProp.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory bus width: " << deviceProp.memoryBusWidth << " bits" << std::endl;
        std::cout << "  Peak memory bandwidth: " << 2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8) / 1.0e6 
                  << " GB/s" << std::endl;
    }

    return 0;
}
